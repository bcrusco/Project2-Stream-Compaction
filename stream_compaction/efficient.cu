#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "common.h"
#include "efficient.h"

namespace StreamCompaction {
namespace Efficient {

__global__ void up_sweep(int n, int d, int *data) {
	int k = threadIdx.x + (blockIdx.x * blockDim.x);

	if (k < n) {
		int p2d = pow(2.0, (double)d);
		int p2da1 = pow(2.0, (double)(d + 1));

		if (k % p2da1 == 0) {
			data[k + p2da1 - 1] += data[k + p2d - 1];
		}
	}	
}

__global__ void down_sweep(int n, int d, int *data) {
	int k = threadIdx.x + (blockIdx.x * blockDim.x);

	if (k < n) {
		int p2d = pow(2.0, (double)d);
		int p2da1 = pow(2.0, (double)(d + 1));

		if (k % p2da1 == 0) {
			int temp = data[k + p2d - 1];
			data[k + p2d - 1] = data[k + p2da1 - 1];
			data[k + p2da1 - 1] += temp;
		}
	}
}

void padArrayRange(int start, int end, int *a) {
	for (int i = start; i < end; i++) {
		a[i] = 0;
	}
}
/**
 * Performs prefix-sum (aka scan) on idata, storing the result into odata.
 */
void scan(int n, int *odata, const int *idata) {
	int m = pow(2, ilog2ceil(n));
	int *new_idata = (int*)malloc(m * sizeof(int));
	dim3 fullBlocksPerGrid((m + blockSize - 1) / blockSize);
	dim3 threadsPerBlock(blockSize);

	// Expand array to next power of 2 size
	for (int i = 0; i < n; i++) {
		new_idata[i] = idata[i];
	}
	padArrayRange(n, m, new_idata);

	// Can use one array for input and output in this implementation
	int *dev_data;
	hipMalloc((void**)&dev_data, m * sizeof(int));
	hipMemcpy(dev_data, new_idata, m * sizeof(int), hipMemcpyHostToDevice);

	// Execute scan on device
	for (int d = 0; d < ilog2ceil(n); d++) {
		up_sweep<<<fullBlocksPerGrid, threadsPerBlock>>>(n, d, dev_data);
	}

	hipMemset((void*)&dev_data[m - 1], 0, sizeof(int));
	for (int d = ilog2ceil(n) - 1; d >= 0; d--) {
		down_sweep<<<fullBlocksPerGrid, threadsPerBlock>>>(n, d, dev_data);
	}

	hipMemcpy(odata, dev_data, n * sizeof(int), hipMemcpyDeviceToHost);

	hipFree(dev_data);
	free(new_idata);
}

/**
 * Performs stream compaction on idata, storing the result into odata.
 * All zeroes are discarded.
 *
 * @param n      The number of elements in idata.
 * @param odata  The array into which to store elements.
 * @param idata  The array of elements to compact.
 * @returns      The number of elements remaining after compaction.
 */
int compact(int n, int *odata, const int *idata) {
	int *bools = (int*)malloc(n * sizeof(int));
	int *scan_data = (int*)malloc(n * sizeof(int));
	int num_remaining = -1;
	dim3 fullBlocksPerGrid((n + blockSize - 1) / blockSize);
	dim3 threadsPerBlock(blockSize);

	int *dev_bools;
	int *dev_idata;
	int *dev_odata;
	int *dev_scan_data;

	hipMalloc((void**)&dev_bools, n * sizeof(int));
	hipMalloc((void**)&dev_idata, n * sizeof(int));
	hipMemcpy(dev_idata, idata, n * sizeof(int), hipMemcpyHostToDevice);

	hipMalloc((void**)&dev_odata, n * sizeof(int));
	hipMalloc((void**)&dev_scan_data, n * sizeof(int));

	// Map to boolean
	StreamCompaction::Common::kernMapToBoolean<<<fullBlocksPerGrid, threadsPerBlock>>>(n, dev_bools, dev_idata);

	hipMemcpy(bools, dev_bools, n * sizeof(int), hipMemcpyDeviceToHost);

	// Execute the scan
	scan(n, scan_data, bools);
	num_remaining = scan_data[n - 1] + bools[n - 1];

	// Execute the scatter
	hipMemcpy(dev_scan_data, scan_data, n * sizeof(int), hipMemcpyHostToDevice);
	StreamCompaction::Common::kernScatter<<<fullBlocksPerGrid, threadsPerBlock>>>(n, dev_odata, dev_idata, dev_bools, dev_scan_data);

	hipMemcpy(odata, dev_odata, n * sizeof(int), hipMemcpyDeviceToHost);

	hipFree(dev_bools);
	hipFree(dev_idata);
	hipFree(dev_odata);
	hipFree(dev_scan_data);
	free(bools);
	free(scan_data);

	return num_remaining;
}

}
}
